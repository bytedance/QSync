#include <torch/extension.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <ATen/cudnn/Handle.h>

#define FatalError(s) do {                                             \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;  \
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(1);                                                           \
} while(0)

#define checkCUDNN(status) do {                                        \
    std::stringstream _error;                                          \
    if (status != HIPDNN_STATUS_SUCCESS) {                              \
      _error << "CUDNN failure: " << hipdnnGetErrorString(status);      \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

torch::Tensor tensor_core_int8_pool(
        torch::Tensor& input, 
        int32_t kernel_size,
        int32_t stride,
        int32_t padding){

    hipdnnHandle_t cudnnHandle = at::native::getCudnnHandle();

    hipdnnTensorDescriptor_t xDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&xDesc));

    int32_t n_in = input.size(0);
    int32_t h_in = input.size(1);
    int32_t w_in = input.size(2);
    int32_t c_in = input.size(3);
    checkCUDNN(hipdnnSetTensor4dDescriptor(xDesc, 
                HIPDNN_TENSOR_NHWC, 
                HIPDNN_DATA_INT8, 
                n_in, c_in, h_in, w_in));

    hipdnnPoolingDescriptor_t poolDesc;
    checkCUDNN(hipdnnCreatePoolingDescriptor(&poolDesc));
    checkCUDNN(hipdnnSetPooling2dDescriptor(poolDesc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN, 
                kernel_size, kernel_size, padding, padding, stride, stride));
    
    int32_t n_out;
    int32_t h_out;
    int32_t w_out;
    int32_t c_out;
    checkCUDNN(hipdnnGetPooling2dForwardOutputDim(poolDesc,xDesc,&n_out,&c_out,&h_out,&w_out));

    hipdnnTensorDescriptor_t yDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&yDesc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(yDesc, 
                HIPDNN_TENSOR_NHWC, 
                HIPDNN_DATA_INT8, 
                n_out, c_out, h_out, w_out));

    auto y = torch::empty({n_out, h_out, w_out, c_out}, torch::dtype(torch::kInt8).device(torch::kCUDA, 0));
   
    float alpha = 1.0;
    float beta = 0.0;

    checkCUDNN(hipdnnPoolingForward(cudnnHandle, poolDesc,
                &alpha, xDesc, input.data<int8_t>(),
                &beta, yDesc, y.data<int8_t>()));

     checkCUDNN(hipdnnDestroyTensorDescriptor(yDesc));
     checkCUDNN(hipdnnDestroyPoolingDescriptor(poolDesc));
     checkCUDNN(hipdnnDestroyTensorDescriptor(xDesc));

     return y;
}

torch::Tensor tensor_core_int8_pool_backward(
        torch::Tensor& y, 
        torch::Tensor& dy, 
        torch::Tensor& x, 
        int32_t kernel_size,
        int32_t stride,
        int32_t padding){
    hipdnnHandle_t cudnnHandle = at::native::getCudnnHandle();

    hipdnnTensorDescriptor_t yDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&yDesc));
    int32_t n_y = y.size(0);
    int32_t h_y = y.size(1);
    int32_t w_y = y.size(2);
    int32_t c_y = y.size(3);
    checkCUDNN(hipdnnSetTensor4dDescriptor(yDesc, 
                HIPDNN_TENSOR_NHWC, 
                HIPDNN_DATA_HALF, 
                n_y, c_y, h_y, w_y));

    hipdnnTensorDescriptor_t xDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&xDesc));
    int32_t n_x= x.size(0);
    int32_t h_x= x.size(1);
    int32_t w_x= x.size(2);
    int32_t c_x= x.size(3);
    checkCUDNN(hipdnnSetTensor4dDescriptor(xDesc, 
                HIPDNN_TENSOR_NHWC, 
                HIPDNN_DATA_HALF, 
                n_x, c_x, h_x, w_x));

    hipdnnPoolingDescriptor_t poolDesc;
    checkCUDNN(hipdnnCreatePoolingDescriptor(&poolDesc));
    checkCUDNN(hipdnnSetPooling2dDescriptor(poolDesc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN, 
                kernel_size, kernel_size, padding, padding, stride, stride));

    auto dx = torch::empty({n_x, h_x, w_x, c_x}, torch::dtype(torch::kHalf).device(torch::kCUDA, 0));

    float alpha = 1.0;
    float beta = 0.0;
    checkCUDNN(hipdnnPoolingBackward(cudnnHandle,poolDesc,
                &alpha, yDesc, 
                y.data<at::Half>(),
                yDesc,
                dy.data<at::Half>(),
                xDesc,
                x.data<at::Half>(),
                &beta,
                xDesc,
                dx.data<at::Half>()));

    checkCUDNN(hipdnnDestroyTensorDescriptor(yDesc));
    checkCUDNN(hipdnnDestroyPoolingDescriptor(poolDesc));
    checkCUDNN(hipdnnDestroyTensorDescriptor(xDesc));

    return dx;
}

torch::Tensor tensor_core_average_pool(
        torch::Tensor& input, 
        int32_t kernel_size,
        int32_t stride,
        int32_t padding){

    hipdnnHandle_t cudnnHandle = at::native::getCudnnHandle();

    hipdnnTensorDescriptor_t xDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&xDesc));

    int32_t n_in = input.size(0);
    int32_t h_in = input.size(1);
    int32_t w_in = input.size(2);
    int32_t c_in = input.size(3);
    checkCUDNN(hipdnnSetTensor4dDescriptor(xDesc, 
                HIPDNN_TENSOR_NHWC, 
                HIPDNN_DATA_HALF, 
                n_in, c_in, h_in, w_in));

    hipdnnPoolingDescriptor_t poolDesc;
    checkCUDNN(hipdnnCreatePoolingDescriptor(&poolDesc));
    checkCUDNN(hipdnnSetPooling2dDescriptor(poolDesc, HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING, HIPDNN_PROPAGATE_NAN, 
                kernel_size, kernel_size, padding, padding, stride, stride));
    
    int32_t n_out;
    int32_t h_out;
    int32_t w_out;
    int32_t c_out;
    checkCUDNN(hipdnnGetPooling2dForwardOutputDim(poolDesc,xDesc,&n_out,&c_out,&h_out,&w_out));

    hipdnnTensorDescriptor_t yDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&yDesc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(yDesc, 
                HIPDNN_TENSOR_NHWC, 
                HIPDNN_DATA_HALF, 
                n_out, c_out, h_out, w_out));

    auto y = torch::empty({n_out, h_out, w_out, c_out}, torch::dtype(torch::kHalf).device(torch::kCUDA, 0));
   
    float alpha = 1.0;
    float beta = 0.0;

    checkCUDNN(hipdnnPoolingForward(cudnnHandle, poolDesc,
                &alpha, xDesc, input.data<at::Half>(),
                &beta, yDesc, y.data<at::Half>()));

     checkCUDNN(hipdnnDestroyTensorDescriptor(yDesc));
     checkCUDNN(hipdnnDestroyPoolingDescriptor(poolDesc));
     checkCUDNN(hipdnnDestroyTensorDescriptor(xDesc));

     return y;
}

torch::Tensor tensor_core_average_pool_backward(
        torch::Tensor& y, 
        torch::Tensor& dy, 
        torch::Tensor& x, 
        int32_t kernel_size,
        int32_t stride,
        int32_t padding){
    hipdnnHandle_t cudnnHandle = at::native::getCudnnHandle();

    hipdnnTensorDescriptor_t yDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&yDesc));
    int32_t n_y = y.size(0);
    int32_t h_y = y.size(1);
    int32_t w_y = y.size(2);
    int32_t c_y = y.size(3);
    checkCUDNN(hipdnnSetTensor4dDescriptor(yDesc, 
                HIPDNN_TENSOR_NHWC, 
                HIPDNN_DATA_HALF, 
                n_y, c_y, h_y, w_y));

    hipdnnTensorDescriptor_t xDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&xDesc));
    int32_t n_x= x.size(0);
    int32_t h_x= x.size(1);
    int32_t w_x= x.size(2);
    int32_t c_x= x.size(3);
    checkCUDNN(hipdnnSetTensor4dDescriptor(xDesc, 
                HIPDNN_TENSOR_NHWC, 
                HIPDNN_DATA_HALF, 
                n_x, c_x, h_x, w_x));

    hipdnnPoolingDescriptor_t poolDesc;
    checkCUDNN(hipdnnCreatePoolingDescriptor(&poolDesc));
    checkCUDNN(hipdnnSetPooling2dDescriptor(poolDesc,HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING, HIPDNN_PROPAGATE_NAN, 
                kernel_size, kernel_size, padding, padding, stride, stride));

    auto dx = torch::empty({n_x, h_x, w_x, c_x}, torch::dtype(torch::kHalf).device(torch::kCUDA, 0));

    float alpha = 1.0;
    float beta = 0.0;
    checkCUDNN(hipdnnPoolingBackward(cudnnHandle,poolDesc,
                &alpha, yDesc, 
                y.data<at::Half>(),
                yDesc,
                dy.data<at::Half>(),
                xDesc,
                x.data<at::Half>(),
                &beta,
                xDesc,
                dx.data<at::Half>()));

    checkCUDNN(hipdnnDestroyTensorDescriptor(yDesc));
    checkCUDNN(hipdnnDestroyPoolingDescriptor(poolDesc));
    checkCUDNN(hipdnnDestroyTensorDescriptor(xDesc));

    return dx;
}
