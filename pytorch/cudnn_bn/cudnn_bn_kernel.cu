#include <torch/extension.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <ATen/cudnn/Handle.h>

#define FatalError(s) do {                                             \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;  \
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(1);                                                           \
} while(0)

#define checkCUDNN(status) do {                                        \
    std::stringstream _error;                                          \
    if (status != HIPDNN_STATUS_SUCCESS) {                              \
      _error << "CUDNN failure: " << hipdnnGetErrorString(status);      \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

#define CUDA_CALL(f) { \
  hipError_t err = (f); \
  if (err != hipSuccess) { \
    std::cout \
        << "    Error occurred: " << err << std::endl; \
    std::exit(1); \
  } \
}


torch::Tensor create_4d_tensor(int64_t N, int64_t H, int64_t W, int64_t C, torch::Dtype OType, torch::Tensor& ref_tensor){
    // auto output = torch::empty({output_size.n(), output_size.h(), output_size.w(), output_size.c()}, torch::dtype(torch::kF32).device(input.device()));
    torch::Tensor output;
    if(ref_tensor.is_contiguous(torch::MemoryFormat::ChannelsLast)){
        output = torch::empty({N, C, H, W}, torch::TensorOptions().dtype(OType).device(ref_tensor.device()).
        memory_format(torch::MemoryFormat::ChannelsLast));
    }else{
        //  create a tensor with nchw meta but NHWC storage
        output = torch::empty({N, H, W, C}, 
        torch::TensorOptions().dtype(OType).device(ref_tensor.device()));
    };
    return output;
}


// https://docs.nvidia.com/deeplearning/cudnn/api/index.html#hipdnnBatchNormalizationForwardTraining
torch::Tensor fp32_bn_cudnn_training(torch::Tensor input, 
        torch::Tensor running_mean, torch::Tensor running_var, 
        torch::Tensor weight, torch::Tensor bias,
        bool training, float exponential_average_factor, float eps)
{
    hipdnnHandle_t cudnnHandle = at::native::getCudnnHandle();
    hipdnnTensorDescriptor_t xDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&xDesc));
    int32_t n_in = input.size(0);
    int32_t c_in = input.size(1);
    int32_t h_in = input.size(2);
    int32_t w_in = input.size(3);

    checkCUDNN(hipdnnSetTensor4dDescriptor(xDesc, 
                HIPDNN_TENSOR_NCHW, 
                HIPDNN_DATA_FLOAT, 
                n_in, c_in, h_in, w_in));
    
    float alpha = 1.0;
    float beta = 0.0;

    hipdnnBatchNormMode_t

    y = torch::empty({n_in, c_in, h_in, w_in}, torch::TensorOptions().dtype(OType).device(ref_tensor.device()));



    checkCUDNN(hipdnnBatchNormalizationForwardTraining(
      hipdnnHandle_t                    handle,
      hipdnnBatchNormMode_t             mode,
      const void                      *alpha,
      const void                      *beta,
      const hipdnnTensorDescriptor_t    xDesc,
      const void                      *x,
      const hipdnnTensorDescriptor_t    yDesc,
      void                            *y,
      const hipdnnTensorDescriptor_t    bnScaleBiasMeanVarDesc,
      const void                      *bnScale,
      const void                      *bnBias,
      double                           exponentialAverageFactor,
      void                            *resultRunningMean,
      void                            *resultRunningVariance,
      double                           epsilon,
      void                            *resultSaveMean,
      void                            *resultSaveInvVariance)
    );

    return y;
}


torch::Tensor tensor_core_fp32_conv(
        torch::Tensor& input, 
        torch::Tensor& weight,
        int32_t stride_h, int32_t stride_w,
        int32_t padding_h, int32_t padding_w,
        int32_t dilation_h, int32_t dilation_w){

    /* only support n_in and c_in multiply of 4 */
    hipdnnHandle_t cudnnHandle = at::native::getCudnnHandle();

    hipdnnTensorDescriptor_t xDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&xDesc));
    int32_t n_in = input.size(0);
    int32_t c_in = input.size(1);
    int32_t h_in = input.size(2);
    int32_t w_in = input.size(3);
    checkCUDNN(hipdnnSetTensor4dDescriptor(xDesc, 
                HIPDNN_TENSOR_NHWC, 
                HIPDNN_DATA_FLOAT, 
                n_in, c_in, h_in, w_in));

    int32_t n_weight= weight.size(0);
    int32_t c_weight = weight.size(1);
    int32_t h_weight = weight.size(2);
    int32_t w_weight = weight.size(3);

    hipdnnFilterDescriptor_t wDesc;
    checkCUDNN(hipdnnCreateFilterDescriptor(&wDesc));
    checkCUDNN(hipdnnSetFilter4dDescriptor(wDesc, 
                HIPDNN_DATA_FLOAT, 
                HIPDNN_TENSOR_NHWC, 
                n_weight, c_weight, h_weight, w_weight));

    hipdnnConvolutionDescriptor_t convDesc;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc, padding_h, padding_w, stride_h, stride_w, dilation_h, dilation_w, 
                HIPDNN_CROSS_CORRELATION,
                HIPDNN_DATA_FLOAT));

    int32_t n_out;
    int32_t h_out;
    int32_t w_out;
    int32_t c_out;
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convDesc, xDesc, wDesc, &n_out, &c_out, &h_out, &w_out));

    hipdnnTensorDescriptor_t yDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&yDesc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(yDesc, 
                HIPDNN_TENSOR_NHWC, 
                HIPDNN_DATA_FLOAT, 
                n_out, c_out, h_out, w_out));

    //std::cout<<"create y tensor"<<std::endl;
    auto y = create_4d_tensor(n_out, h_out, w_out, c_out, torch::kF32, input);

    //hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
    hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
   
    float alpha = 1.0;
    //float alpha = 1;
    float beta = 0.0;

    //size_t ws_size = 355968;
    size_t ws_size;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,xDesc,wDesc,convDesc,yDesc,algo,&ws_size));
    auto workspace = torch::empty({static_cast<int64_t>(ws_size)}, torch::dtype(torch::kInt32).device(input.device()));

    checkCUDNN(hipdnnConvolutionForward(cudnnHandle,
                &alpha,xDesc,input.data_ptr<float>(),
                wDesc,weight.data_ptr<float>(),
                convDesc,
                algo,
                workspace.data_ptr<int32_t>(),
                ws_size,
                &beta,yDesc,
                y.data_ptr<float>()));

     checkCUDNN(hipdnnDestroyTensorDescriptor(yDesc));
     checkCUDNN(hipdnnDestroyConvolutionDescriptor(convDesc));
     checkCUDNN(hipdnnDestroyFilterDescriptor(wDesc));
     checkCUDNN(hipdnnDestroyTensorDescriptor(xDesc));
     return y;
}


torch::Tensor tensor_core_fp32_conv_nhwc(
        torch::Tensor& input, 
        torch::Tensor& weight,
        int32_t stride_h, int32_t stride_w,
        int32_t padding_h, int32_t padding_w,
        int32_t dilation_h, int32_t dilation_w){

    /* only support n_in and c_in multiply of 4 */
    hipdnnHandle_t cudnnHandle = at::native::getCudnnHandle();

    hipdnnTensorDescriptor_t xDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&xDesc));
    int32_t n_in = input.size(0);
    int32_t h_in = input.size(1);
    int32_t w_in = input.size(2);
    int32_t c_in = input.size(3);
    checkCUDNN(hipdnnSetTensor4dDescriptor(xDesc, 
                HIPDNN_TENSOR_NHWC, 
                HIPDNN_DATA_FLOAT, 
                n_in, c_in, h_in, w_in));

    int32_t n_weight=  weight.size(0);
    int32_t h_weight = weight.size(1);
    int32_t w_weight = weight.size(2);
    int32_t c_weight = weight.size(3);

    hipdnnFilterDescriptor_t wDesc;
    checkCUDNN(hipdnnCreateFilterDescriptor(&wDesc));
    checkCUDNN(hipdnnSetFilter4dDescriptor(wDesc, 
                HIPDNN_DATA_FLOAT, 
                HIPDNN_TENSOR_NHWC, 
                n_weight, c_weight, h_weight, w_weight));

    hipdnnConvolutionDescriptor_t convDesc;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc, padding_h, padding_w, stride_h, stride_w, dilation_h, dilation_w, 
                HIPDNN_CROSS_CORRELATION,
                HIPDNN_DATA_FLOAT));

    int32_t n_out;
    int32_t h_out;
    int32_t w_out;
    int32_t c_out;
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convDesc, xDesc, wDesc, &n_out, &c_out, &h_out, &w_out));

    hipdnnTensorDescriptor_t yDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&yDesc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(yDesc, 
                HIPDNN_TENSOR_NHWC, 
                HIPDNN_DATA_FLOAT, 
                n_out, c_out, h_out, w_out));

    //std::cout<<"create y tensor"<<std::endl;
    auto y = create_4d_tensor(n_out, h_out, w_out, c_out, torch::kF32, input);

    //hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
    hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
   
    float alpha = 1.0;
    //float alpha = 1;
    float beta = 0.0;

    //size_t ws_size = 355968;
    size_t ws_size;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,xDesc,wDesc,convDesc,yDesc,algo,&ws_size));
    auto workspace = torch::empty({static_cast<int64_t>(ws_size)}, torch::dtype(torch::kInt32).device(input.device()));

    checkCUDNN(hipdnnConvolutionForward(cudnnHandle,
                &alpha,xDesc,input.data_ptr<float>(),
                wDesc,weight.data_ptr<float>(),
                convDesc,
                algo,
                workspace.data_ptr<int32_t>(),
                ws_size,
                &beta,yDesc,
                y.data_ptr<float>()));

     checkCUDNN(hipdnnDestroyTensorDescriptor(yDesc));
     checkCUDNN(hipdnnDestroyConvolutionDescriptor(convDesc));
     checkCUDNN(hipdnnDestroyFilterDescriptor(wDesc));
     checkCUDNN(hipdnnDestroyTensorDescriptor(xDesc));
     return y;
}
